#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall(err)					__cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError(err, errMessage)		__cudaCheckError(err, errMessage)
#define CudaSyncAndCheckError(errMessage)	__cudaSyncAndCheckError(errMessage)

__host__ inline std::string __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	if (hipSuccess != err)
		return hipGetErrorString(err);
	else
#endif //CUDA_ERROR_CHECK
		return "";
}

__host__ inline bool __cudaCheckError(hipError_t err, const char *errorMessage)
{
#ifdef CUDA_ERROR_CHECK
	if (err != hipSuccess)
	{
		auto errorMsgChar = hipGetErrorString(err);
		std::string errorMsg{ errorMsgChar };

		std::memcpy((void *)errorMessage, errorMsgChar, errorMsg.length());
		std::memset((void *)&errorMessage[errorMsg.length()], '\0', 1ull);

		return false;
	}
#endif //CUDA_ERROR_CHECK

	return true;
}

__host__ inline bool __cudaSyncAndCheckError(const char *errorMessage)
{
	hipError_t response{ hipSuccess };
	std::string cudaErrors{ "" };

#ifdef CUDA_ERROR_CHECK
	response = hipGetLastError();
	if (response != hipSuccess)
	{
		while (response != hipSuccess)
		{
			if (!cudaErrors.empty()) cudaErrors += " <- ";
			cudaErrors += hipGetErrorString(response);
			response = hipGetLastError();
		}
		auto errorChar = cudaErrors.c_str();

		std::memcpy((void *)errorMessage, errorChar, cudaErrors.length());
		std::memset((void *)&errorMessage[cudaErrors.length()], '\0', 1ull);

		return false;
	}
#endif //CUDA_ERROR_CHECK

	response = hipDeviceSynchronize();

	if (response != hipSuccess)
	{
		response = hipGetLastError();

		while (response != hipSuccess)
		{
			if (!cudaErrors.empty()) cudaErrors += " <- ";
			cudaErrors += hipGetErrorString(response);
			response = hipGetLastError();
		}
		auto errorChar = cudaErrors.c_str();

		std::memcpy((void *)errorMessage, errorChar, cudaErrors.length());
		std::memset((void *)&errorMessage[cudaErrors.length()], '\0', 1ull);

		return false;
	}
	return true;
}
